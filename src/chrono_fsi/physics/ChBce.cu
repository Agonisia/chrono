#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Arman Pazouki, Milad Rakhsha, Wei Hu
// =============================================================================
//
// Base class for processing boundary condition enforcing (bce) markers forces
// in FSI system.
// =============================================================================

#include "chrono_fsi/physics/ChBce.cuh"
#include "chrono_fsi/physics/ChSphGeneral.cuh"
#include <type_traits>

namespace chrono {
namespace fsi {

// -----------------------------------------------------------------------------

__device__ double atomicAdd_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

// -----------------------------------------------------------------------------

__global__ void Populate_RigidSPH_MeshPos_LRF_D(Real3* rigid_BCEcoords_D,
                                                Real4* posRadD,
                                                uint* rigid_BCEsolids_D,
                                                Real3* posRigidD,
                                                Real4* qD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numRigidMarkers)
        return;

    int rigidIndex = rigid_BCEsolids_D[index];
    uint rigidMarkerIndex = index + numObjectsD.startRigidMarkers;
    Real4 q4 = qD[rigidIndex];
    Real3 a1, a2, a3;
    RotationMatirixFromQuaternion(a1, a2, a3, q4);
    Real3 dist3 = mR3(posRadD[rigidMarkerIndex]) - posRigidD[rigidIndex];
    Real3 dist3LF = InverseRotate_By_RotationMatrix_DeviceHost(a1, a2, a3, dist3);

    // Save the coordinates in the local reference of a rigid body
    rigid_BCEcoords_D[index] = dist3LF;
}

// -----------------------------------------------------------------------------

__global__ void CalcRigidForces_D(Real3* rigid_FSI_ForcesD,
                                  Real3* rigid_FSI_TorquesD,
                                  Real4* derivVelRhoD,
                                  Real4* derivVelRhoD_old,
                                  Real4* posRadD,
                                  uint* rigid_BCEsolids_D,
                                  Real3* posRigidD,
                                  Real3* rigid_BCEcoords_D) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numRigidMarkers)
        return;

    int RigidIndex = rigid_BCEsolids_D[index];
    uint rigidMarkerIndex = index + numObjectsD.startRigidMarkers;
    Real3 Force = (mR3(derivVelRhoD[rigidMarkerIndex]) * paramsD.Beta +
                   mR3(derivVelRhoD_old[rigidMarkerIndex]) * (1 - paramsD.Beta)) *
                  paramsD.markerMass;

    if (std::is_same<Real, double>::value) {
        atomicAdd_double((double*)&(rigid_FSI_ForcesD[RigidIndex].x), Force.x);
        atomicAdd_double((double*)&(rigid_FSI_ForcesD[RigidIndex].y), Force.y);
        atomicAdd_double((double*)&(rigid_FSI_ForcesD[RigidIndex].z), Force.z);
    } else {
        atomicAdd((float*)&(rigid_FSI_ForcesD[RigidIndex].x), Force.x);
        atomicAdd((float*)&(rigid_FSI_ForcesD[RigidIndex].y), Force.y);
        atomicAdd((float*)&(rigid_FSI_ForcesD[RigidIndex].z), Force.z);
    }
    Real3 dist3 = Distance(mR3(posRadD[rigidMarkerIndex]), posRigidD[RigidIndex]);
    Real3 mtorque = cross(dist3, Force);

    if (std::is_same<Real, double>::value) {
        atomicAdd_double((double*)&(rigid_FSI_TorquesD[RigidIndex].x), mtorque.x);
        atomicAdd_double((double*)&(rigid_FSI_TorquesD[RigidIndex].y), mtorque.y);
        atomicAdd_double((double*)&(rigid_FSI_TorquesD[RigidIndex].z), mtorque.z);
    } else {
        atomicAdd((float*)&(rigid_FSI_TorquesD[RigidIndex].x), mtorque.x);
        atomicAdd((float*)&(rigid_FSI_TorquesD[RigidIndex].y), mtorque.y);
        atomicAdd((float*)&(rigid_FSI_TorquesD[RigidIndex].z), mtorque.z);
    }
}

__global__ void CalcFlex1DForces_D(Real3* flex1D_FSIforces_D,  // FEA node forces (output)
                                   Real4* derivVelRhoD,        // dv/dt
                                   Real4* derivVelRhoD_old,    // dv/dt
                                   uint2* flex1D_Nodes_D,      // segment node indices
                                   uint3* flex1D_BCEsolids_D,  // association of flex BCEs with a mesh and segment
                                   Real3* flex1D_BCEcoords_D   // local coordinates of BCE markers on FEA 1-D segments
) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numFlexMarkers1D)
        return;

    uint flex_index = index + numObjectsD.startFlexMarkers1D;  // index for current 1-D flex BCE marker
    uint3 flex_solid = flex1D_BCEsolids_D[index];              // associated flex mesh and segment
    ////uint flex_mesh = flex_solid.x;                             // index of associated mesh
    ////uint flex_mesh_seg = flex_solid.y;                         // index of segment in associated mesh
    uint flex_seg = flex_solid.z;                              // index of segment in global list

    // Fluid force on BCE marker
    Real3 Force =
        (mR3(derivVelRhoD[flex_index]) * paramsD.Beta + mR3(derivVelRhoD_old[flex_index]) * (1 - paramsD.Beta)) *
        paramsD.markerMass;

    uint2 seg_nodes = flex1D_Nodes_D[flex_seg];  // indices of the 2 nodes on associated segment
    uint n0 = seg_nodes.x;
    uint n1 = seg_nodes.y;

    Real lambda0 = flex1D_BCEcoords_D[index].x;  // segment coordinate
    Real lambda1 = 1 - lambda0;                  // segment coordinate

    // Split BCE marker force to the 2 nodes of the 1-D segment and accumulate
    if (std::is_same<Real, double>::value) {
        atomicAdd_double((double*)&(flex1D_FSIforces_D[n0].x), lambda0 * Force.x);
        atomicAdd_double((double*)&(flex1D_FSIforces_D[n0].y), lambda0 * Force.y);
        atomicAdd_double((double*)&(flex1D_FSIforces_D[n0].z), lambda0 * Force.z);

        atomicAdd_double((double*)&(flex1D_FSIforces_D[n1].x), lambda1 * Force.x);
        atomicAdd_double((double*)&(flex1D_FSIforces_D[n1].y), lambda1 * Force.y);
        atomicAdd_double((double*)&(flex1D_FSIforces_D[n1].z), lambda1 * Force.z);
    } else {
        atomicAdd((float*)&(flex1D_FSIforces_D[n0].x), lambda0 * Force.x);
        atomicAdd((float*)&(flex1D_FSIforces_D[n0].y), lambda0 * Force.y);
        atomicAdd((float*)&(flex1D_FSIforces_D[n0].z), lambda0 * Force.z);

        atomicAdd((float*)&(flex1D_FSIforces_D[n1].x), lambda1 * Force.x);
        atomicAdd((float*)&(flex1D_FSIforces_D[n1].y), lambda1 * Force.y);
        atomicAdd((float*)&(flex1D_FSIforces_D[n1].z), lambda1 * Force.z);
    }
}

__global__ void CalcFlex2DForces_D(Real3* flex2D_FSIforces_D,  // FEA node forces (output)
                                   Real4* derivVelRhoD,        // dv/dt
                                   Real4* derivVelRhoD_old,    // dv/dt
                                   uint3* flex2D_Nodes_D,      // triangle node indices
                                   uint3* flex2D_BCEsolids_D,  // association of flex BCEs with a mesh and face
                                   Real3* flex2D_BCEcoords_D   // local coordinates of BCE markers on FEA 2-D faces
) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numFlexMarkers2D)
        return;

    uint flex_index = index + numObjectsD.startFlexMarkers2D;  // index for current 2-D flex BCE marker
    uint3 flex_solid = flex2D_BCEsolids_D[index];              // associated flex mesh and face
    ////uint flex_mesh = flex_solid.x;                             // index of associated mesh
    ////uint flex_mesh_tri = flex_solid.y;                         // index of triangle in associated mesh
    uint flex_tri = flex_solid.z;                              // index of triangle in global list

    // Fluid force on BCE marker
    Real3 Force =
        (mR3(derivVelRhoD[flex_index]) * paramsD.Beta + mR3(derivVelRhoD_old[flex_index]) * (1 - paramsD.Beta)) *
        paramsD.markerMass;

    auto tri_nodes = flex2D_Nodes_D[flex_tri];  // indices of the 3 nodes on associated face
    uint n0 = tri_nodes.x;
    uint n1 = tri_nodes.y;
    uint n2 = tri_nodes.z;

    Real lambda0 = flex2D_BCEcoords_D[index].x;  // barycentric coordinate
    Real lambda1 = flex2D_BCEcoords_D[index].y;  // barycentric coordinate
    Real lambda2 = 1 - lambda0 - lambda1;        // barycentric coordinate

    // Split BCE marker force to the 3 nodes of the 2-D face and accumulate
    if (std::is_same<Real, double>::value) {
        atomicAdd_double((double*)&(flex2D_FSIforces_D[n0].x), lambda0 * Force.x);
        atomicAdd_double((double*)&(flex2D_FSIforces_D[n0].y), lambda0 * Force.y);
        atomicAdd_double((double*)&(flex2D_FSIforces_D[n0].z), lambda0 * Force.z);

        atomicAdd_double((double*)&(flex2D_FSIforces_D[n1].x), lambda1 * Force.x);
        atomicAdd_double((double*)&(flex2D_FSIforces_D[n1].y), lambda1 * Force.y);
        atomicAdd_double((double*)&(flex2D_FSIforces_D[n1].z), lambda1 * Force.z);

        atomicAdd_double((double*)&(flex2D_FSIforces_D[n2].x), lambda2 * Force.x);
        atomicAdd_double((double*)&(flex2D_FSIforces_D[n2].y), lambda2 * Force.y);
        atomicAdd_double((double*)&(flex2D_FSIforces_D[n2].z), lambda2 * Force.z);
    } else {
        atomicAdd((float*)&(flex2D_FSIforces_D[n0].x), lambda0 * Force.x);
        atomicAdd((float*)&(flex2D_FSIforces_D[n0].y), lambda0 * Force.y);
        atomicAdd((float*)&(flex2D_FSIforces_D[n0].z), lambda0 * Force.z);

        atomicAdd((float*)&(flex2D_FSIforces_D[n1].x), lambda1 * Force.x);
        atomicAdd((float*)&(flex2D_FSIforces_D[n1].y), lambda1 * Force.y);
        atomicAdd((float*)&(flex2D_FSIforces_D[n1].z), lambda1 * Force.z);

        atomicAdd((float*)&(flex2D_FSIforces_D[n2].x), lambda2 * Force.x);
        atomicAdd((float*)&(flex2D_FSIforces_D[n2].y), lambda2 * Force.y);
        atomicAdd((float*)&(flex2D_FSIforces_D[n2].z), lambda2 * Force.z);
    }
}

// -----------------------------------------------------------------------------

__device__ void BCE_modification_Share(Real3& sumVW,
                                       Real3& sumRhoRW,
                                       Real& sumPW,
                                       Real3& sumTauXxYyZzW,
                                       Real3& sumTauXyXzYzW,
                                       Real& sumWFluid,
                                       int& isAffectedV,
                                       int& isAffectedP,
                                       int3 gridPos,
                                       Real3 posRadA,
                                       Real4* sortedPosRad,
                                       Real3* sortedVelMas,
                                       Real4* sortedRhoPreMu,
                                       Real3* sortedTauXxYyZz,
                                       Real3* sortedTauXyXzYz,
                                       uint* cellStart,
                                       uint* cellEnd) {
    uint gridHash = calcGridHash(gridPos);
    // get start of bucket for this cell
    uint startIndex = cellStart[gridHash];
    uint endIndex = cellEnd[gridHash];

    for (uint j = startIndex; j < endIndex; j++) {
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 dist3 = Distance(posRadA, posRadB);
        Real dd = dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z;
        Real4 rhoPresMuB = sortedRhoPreMu[j];
        Real kernel_radius = RESOLUTION_LENGTH_MULT * paramsD.HSML;
        if (dd > kernel_radius * kernel_radius || rhoPresMuB.w > -0.5)
            continue;
        Real d = length(dist3);
        Real Wd = W3h(d, sortedPosRad[j].w);
        Real3 velMasB = sortedVelMas[j];
        sumVW += velMasB * Wd;
        sumRhoRW += rhoPresMuB.x * dist3 * Wd;
        sumPW += rhoPresMuB.y * Wd;
        sumWFluid += Wd;
        sumTauXxYyZzW += sortedTauXxYyZz[j] * Wd;
        sumTauXyXzYzW += sortedTauXyXzYz[j] * Wd;
    }
}

__global__ void BCE_VelocityPressureStress(Real3* velMas_ModifiedBCE,
                                           Real4* rhoPreMu_ModifiedBCE,
                                           Real3* tauXxYyZz_ModifiedBCE,
                                           Real3* tauXyXzYz_ModifiedBCE,
                                           Real4* sortedPosRad,
                                           Real3* sortedVelMas,
                                           Real4* sortedRhoPreMu,
                                           Real3* sortedTauXxYyZz,
                                           Real3* sortedTauXyXzYz,
                                           uint* cellStart,
                                           uint* cellEnd,
                                           uint* mapOriginalToSorted,
                                           uint* extendedActivityIdD,
                                           Real3* bceAcc,
                                           int2 newPortion,
                                           volatile bool* isErrorD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    uint sphIndex = index + newPortion.x;
    if (index >= newPortion.y - newPortion.x)
        return;

    // no need to do anything if it is not an active particle
    uint originalIndex = sphIndex;
    uint activity = extendedActivityIdD[originalIndex];
    if (activity == 0)
        return;

    uint bceIndex = index;
    if (paramsD.bceTypeWall == BceVersion::ORIGINAL)
        bceIndex = index + numObjectsD.numBoundaryMarkers;

    uint idA = mapOriginalToSorted[sphIndex];

    Real4 rhoPreMuA = sortedRhoPreMu[idA];
    Real3 posRadA = mR3(sortedPosRad[idA]);
    Real3 velMasA = sortedVelMas[idA];
    int isAffectedV = 0;
    int isAffectedP = 0;

    Real3 sumVW = mR3(0);
    Real3 sumRhoRW = mR3(0);
    Real sumPW = 0;
    Real sumWFluid = 0;
    Real3 sumTauXxYyZzW = mR3(0);
    Real3 sumTauXyXzYzW = mR3(0);

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);

    // examine neighbouring cells
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                BCE_modification_Share(sumVW, sumRhoRW, sumPW, sumTauXxYyZzW, sumTauXyXzYzW, sumWFluid, isAffectedV,
                                       isAffectedP, neighbourPos, posRadA, sortedPosRad, sortedVelMas, sortedRhoPreMu,
                                       sortedTauXxYyZz, sortedTauXyXzYz, cellStart, cellEnd);
            }
        }
    }

    if (abs(sumWFluid) > EPSILON) {
        // modify velocity
        Real3 modifiedBCE_v = 2 * velMasA - sumVW / sumWFluid;
        velMas_ModifiedBCE[bceIndex] = modifiedBCE_v;

        // modify pressure and stress
        Real3 aW = mR3(0.0);
        if (rhoPreMuA.w > 0.5 && rhoPreMuA.w < 1.5) {  // type = 1
            int rigidBceIndex = sphIndex - numObjectsD.startRigidMarkers;
            if (rigidBceIndex < 0 || rigidBceIndex >= numObjectsD.numRigidMarkers) {
                printf("Error! particle index out of bound: thrown from ChBce.cu, BCE_VelocityPressureStress !\n");
                *isErrorD = true;
                return;
            }
            // rigid BCE marker acceleration
            aW = bceAcc[rigidBceIndex];
        }
        if (rhoPreMuA.w > 1.5 && rhoPreMuA.w < 2.5) {  // type = 2
            int flexBceIndex = sphIndex - numObjectsD.startFlexMarkers1D;
            if (flexBceIndex < 0 || flexBceIndex >= numObjectsD.numFlexMarkers1D) {
                printf("Error! particle index out of bound: thrown from ChBce.cu, BCE_VelocityPressureStress !\n");
                *isErrorD = true;
                return;
            }
            // flex1D BCE marker acceleration
            aW = bceAcc[flexBceIndex + numObjectsD.numRigidMarkers];
        }
        if (rhoPreMuA.w > 2.5 && rhoPreMuA.w < 3.5) {  // type = 3
            int flexBceIndex = sphIndex - numObjectsD.startFlexMarkers2D;
            if (flexBceIndex < 0 || flexBceIndex >= numObjectsD.numFlexMarkers2D) {
                printf("Error! particle index out of bound: thrown from ChBce.cu, BCE_VelocityPressureStress !\n");
                *isErrorD = true;
                return;
            }
            // flex2D BCE marker acceleration
            aW = bceAcc[flexBceIndex + numObjectsD.numRigidMarkers + numObjectsD.numFlexMarkers1D];
        }

        Real pressure = (sumPW + dot(paramsD.gravity - aW, sumRhoRW)) / sumWFluid;
        Real density = InvEos(pressure);
        rhoPreMu_ModifiedBCE[bceIndex] = mR4(density, pressure, rhoPreMuA.z, rhoPreMuA.w);
        if (paramsD.elastic_SPH) {
            Real3 tauXxYyZz = (sumTauXxYyZzW + dot(paramsD.gravity - aW, sumRhoRW)) / sumWFluid;
            Real3 tauXyXzYz = sumTauXyXzYzW / sumWFluid;
            tauXxYyZz_ModifiedBCE[bceIndex] = mR3(tauXxYyZz.x, tauXxYyZz.y, tauXxYyZz.z);
            tauXyXzYz_ModifiedBCE[bceIndex] = mR3(tauXyXzYz.x, tauXyXzYz.y, tauXyXzYz.z);
        }
    } else {
        rhoPreMu_ModifiedBCE[bceIndex] = mR4(paramsD.rho0, paramsD.BASEPRES, paramsD.mu0, rhoPreMuA.w);
        velMas_ModifiedBCE[bceIndex] = mR3(0.0);
        if (paramsD.elastic_SPH) {
            tauXxYyZz_ModifiedBCE[bceIndex] = mR3(0.0);
            tauXyXzYz_ModifiedBCE[bceIndex] = mR3(0.0);
        }
    }

    sortedVelMas[idA] = velMas_ModifiedBCE[bceIndex];
    sortedRhoPreMu[idA] = rhoPreMu_ModifiedBCE[bceIndex];
    if (paramsD.elastic_SPH) {
        sortedTauXxYyZz[idA] = tauXxYyZz_ModifiedBCE[bceIndex];
        sortedTauXyXzYz[idA] = tauXyXzYz_ModifiedBCE[bceIndex];
    }
}

// -----------------------------------------------------------------------------

__global__ void CalcRigidBceAccelerationD(Real3* bceAcc,
                                          Real4* q_fsiBodies_D,
                                          Real3* accRigid_fsiBodies_D,
                                          Real3* omegaVelLRF_fsiBodies_D,
                                          Real3* omegaAccLRF_fsiBodies_D,
                                          Real3* rigid_BCEcoords_D,
                                          const uint* rigid_BCEsolids_D) {
    uint bceIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (bceIndex >= numObjectsD.numRigidMarkers)
        return;

    int rigidBodyIndex = rigid_BCEsolids_D[bceIndex];

    // linear acceleration (CM)
    Real3 acc3 = accRigid_fsiBodies_D[rigidBodyIndex];

    Real4 q4 = q_fsiBodies_D[rigidBodyIndex];
    Real3 a1, a2, a3;
    RotationMatirixFromQuaternion(a1, a2, a3, q4);
    Real3 wVel3 = omegaVelLRF_fsiBodies_D[rigidBodyIndex];
    Real3 rigidSPH_MeshPos_LRF = rigid_BCEcoords_D[bceIndex];
    Real3 wVelCrossS = cross(wVel3, rigidSPH_MeshPos_LRF);
    Real3 wVelCrossWVelCrossS = cross(wVel3, wVelCrossS);

    // centrigugal acceleration
    acc3 += mR3(dot(a1, wVelCrossWVelCrossS), dot(a2, wVelCrossWVelCrossS), dot(a3, wVelCrossWVelCrossS));

    Real3 wAcc3 = omegaAccLRF_fsiBodies_D[rigidBodyIndex];
    Real3 wAccCrossS = cross(wAcc3, rigidSPH_MeshPos_LRF);

    // tangential acceleration
    acc3 += mR3(dot(a1, wAccCrossS), dot(a2, wAccCrossS), dot(a3, wAccCrossS));

    bceAcc[bceIndex] = acc3;
}

__global__ void CalcMeshMarker1DAcceleration_D(
    Real3* bceAcc,              // marker accelerations (output)
    Real3* acc_fsi_fea_D,       // accelerations of FEA 1-D segment nodes
    uint2* flex1D_Nodes_D,      // segment node indices
    uint3* flex1D_BCEsolids_D,  // association of flex BCEs with a mesh and segment
    Real3* flex1D_BCEcoords_D   // local coordinates of BCE markers on FEA 1-D segments
) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numFlexMarkers1D)
        return;

    uint flex_index = index + numObjectsD.startFlexMarkers1D;  // index for current 1-D flex BCE marker
    uint3 flex_solid = flex1D_BCEsolids_D[index];              // associated flex mesh and segment
    ////uint flex_mesh = flex_solid.x;                             // index of associated mesh
    ////uint flex_mesh_seg = flex_solid.y;                         // index of segment in associated mesh
    uint flex_seg = flex_solid.z;                              // index of segment in global list

    uint2 seg_nodes = flex1D_Nodes_D[flex_seg];  // indices of the 2 nodes on associated segment
    Real3 A0 = acc_fsi_fea_D[seg_nodes.x];       // (absolute) acceleration of node 0
    Real3 A1 = acc_fsi_fea_D[seg_nodes.y];       // (absolute) acceleration of node 1

    Real lambda0 = flex1D_BCEcoords_D[index].x;  // segment coordinate
    Real lambda1 = 1 - lambda0;                  // segment coordinate

    bceAcc[flex_index] = A0 * lambda0 + A1 * lambda1;
}

__global__ void CalcMeshMarker2DAcceleration_D(
    Real3* bceAcc,              // marker accelerations (output)
    Real3* acc_fsi_fea_D,       // accelerations of FEA 2-D face nodes
    uint3* flex2D_Nodes_D,      // triangle node indices
    uint3* flex2D_BCEsolids_D,  // association of flex BCEs with a mesh and face
    Real3* flex2D_BCEcoords_D   // local coordinates of BCE markers on FEA 2-D faces
) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numFlexMarkers2D)
        return;

    uint flex_index = index + numObjectsD.startFlexMarkers2D;  // index for current 2-D flex BCE marker
    uint3 flex_solid = flex2D_BCEsolids_D[index];              // associated flex mesh and face
    ////uint flex_mesh = flex_solid.x;                             // index of associated mesh
    ////uint flex_mesh_tri = flex_solid.y;                         // index of triangle in associated mesh
    uint flex_tri = flex_solid.z;                              // index of triangle in global list

    auto tri_nodes = flex2D_Nodes_D[flex_tri];  // indices of the 3 nodes on associated face
    Real3 A0 = acc_fsi_fea_D[tri_nodes.x];      // (absolute) acceleration of node 0
    Real3 A1 = acc_fsi_fea_D[tri_nodes.y];      // (absolute) acceleration of node 1
    Real3 A2 = acc_fsi_fea_D[tri_nodes.z];      // (absolute) acceleration of node 2

    Real lambda0 = flex2D_BCEcoords_D[index].x;  // barycentric coordinate
    Real lambda1 = flex2D_BCEcoords_D[index].y;  // barycentric coordinate
    Real lambda2 = 1 - lambda0 - lambda1;        // barycentric coordinate

    bceAcc[flex_index] = A0 * lambda0 + A1 * lambda1 + A2 * lambda2;
}

// -----------------------------------------------------------------------------

__global__ void UpdateBodyMarkerStateD(Real4* posRadD,
                                       Real3* velMasD,
                                       Real3* rigid_BCEcoords_D,
                                       uint* rigid_BCEsolids_D,
                                       Real3* posRigidD,
                                       Real4* velMassRigidD,
                                       Real3* omegaLRF_D,
                                       Real4* qD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numRigidMarkers)
        return;

    uint rigidMarkerIndex = index + numObjectsD.startRigidMarkers;
    int rigidBodyIndex = rigid_BCEsolids_D[index];

    Real4 q4 = qD[rigidBodyIndex];
    Real3 a1, a2, a3;
    RotationMatirixFromQuaternion(a1, a2, a3, q4);

    Real3 rigidSPH_MeshPos_LRF = rigid_BCEcoords_D[index];

    // position
    Real h = posRadD[rigidMarkerIndex].w;
    Real3 p_Rigid = posRigidD[rigidBodyIndex];
    Real3 pos =
        p_Rigid + mR3(dot(a1, rigidSPH_MeshPos_LRF), dot(a2, rigidSPH_MeshPos_LRF), dot(a3, rigidSPH_MeshPos_LRF));
    posRadD[rigidMarkerIndex] = mR4(pos, h);

    // velocity
    Real4 vM_Rigid = velMassRigidD[rigidBodyIndex];
    Real3 omega3 = omegaLRF_D[rigidBodyIndex];
    Real3 omegaCrossS = cross(omega3, rigidSPH_MeshPos_LRF);
    velMasD[rigidMarkerIndex] = mR3(vM_Rigid) + mR3(dot(a1, omegaCrossS), dot(a2, omegaCrossS), dot(a3, omegaCrossS));
}

__global__ void UpdateMeshMarker1DState_D(
    Real4* posRadD,             // marker positions (output)
    Real3* velMasD,             // marker velocities (output)
    Real3* pos_fsi_fea_D,       // positions of FEA 1-D segment nodes
    Real3* vel_fsi_fea_D,       // velocities of FEA 1-D segment nodes
    uint2* flex1D_Nodes_D,      // segment node indices
    uint3* flex1D_BCEsolids_D,  // association of flex BCEs with a mesh and segment
    Real3* flex1D_BCEcoords_D   // local coordinates of BCE markers on FEA 1-D segments
) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numFlexMarkers1D)
        return;

    uint flex_index = index + numObjectsD.startFlexMarkers1D;  // index for current 1-D flex BCE marker
    uint3 flex_solid = flex1D_BCEsolids_D[index];              // associated flex mesh and segment
    ////uint flex_mesh = flex_solid.x;                             // index of associated mesh
    ////uint flex_mesh_seg = flex_solid.y;                         // index of segment in associated mesh
    uint flex_seg = flex_solid.z;                              // index of segment in global list

    uint2 seg_nodes = flex1D_Nodes_D[flex_seg];  // indices of the 2 nodes on associated segment
    Real3 P0 = pos_fsi_fea_D[seg_nodes.x];       // (absolute) position of node 0
    Real3 P1 = pos_fsi_fea_D[seg_nodes.y];       // (absolute) position of node 1
    Real3 V0 = vel_fsi_fea_D[seg_nodes.x];       // (absolute) velocity of node 0
    Real3 V1 = vel_fsi_fea_D[seg_nodes.y];       // (absolute) velocity of node 1

    Real3 x_dir = normalize(P1 - P0);
    Real3 y_dir = mR3(-x_dir.y - x_dir.z, x_dir.x - x_dir.z, x_dir.x + x_dir.y);
    y_dir = y_dir / length(y_dir);
    Real3 z_dir = cross(x_dir, y_dir);

    Real lambda0 = flex1D_BCEcoords_D[index].x;  // segment coordinate
    Real lambda1 = 1 - lambda0;                  // segment coordinate
    Real y_val = flex1D_BCEcoords_D[index].y;    // off-segment y coordinate
    Real z_val = flex1D_BCEcoords_D[index].z;    // off-segment z coordinate

    Real3 P = P0 * lambda0 + P1 * lambda1 + y_val * y_dir + z_val * z_dir;  // BCE marker position
    Real3 V = V0 * lambda0 + V1 * lambda1;                                  // BCE marker velocity

    Real h = posRadD[flex_index].w;
    posRadD[flex_index] = mR4(P, h);
    velMasD[flex_index] = V;
}

__global__ void UpdateMeshMarker2DState_D(
    Real4* posRadD,             // marker positions (output)
    Real3* velMasD,             // marker velocities (output)
    Real3* pos_fsi_fea_D,       // positions of FEA 2-D face nodes
    Real3* vel_fsi_fea_D,       // velocities of FEA 2-D face nodes
    uint3* flex2D_Nodes_D,      // triangle node indices
    uint3* flex2D_BCEsolids_D,  // association of flex BCEs with a mesh and face
    Real3* flex2D_BCEcoords_D   // local coordinates of BCE markers on FEA 2-D faces
) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numFlexMarkers2D)
        return;

    uint flex_index = index + numObjectsD.startFlexMarkers2D;  // index for current 2-D flex BCE marker
    uint3 flex_solid = flex2D_BCEsolids_D[index];              // associated flex mesh and face
    ////uint flex_mesh = flex_solid.x;                             // index of associated mesh
    ////uint flex_mesh_tri = flex_solid.y;                         // index of triangle in associated mesh
    uint flex_tri = flex_solid.z;                              // index of triangle in global list

    auto tri_nodes = flex2D_Nodes_D[flex_tri];  // indices of the 3 nodes on associated face
    Real3 P0 = pos_fsi_fea_D[tri_nodes.x];      // (absolute) position of node 0
    Real3 P1 = pos_fsi_fea_D[tri_nodes.y];      // (absolute) position of node 1
    Real3 P2 = pos_fsi_fea_D[tri_nodes.z];      // (absolute) position of node 2
    Real3 V0 = vel_fsi_fea_D[tri_nodes.x];      // (absolute) velocity of node 0
    Real3 V1 = vel_fsi_fea_D[tri_nodes.y];      // (absolute) velocity of node 1
    Real3 V2 = vel_fsi_fea_D[tri_nodes.z];      // (absolute) velocity of node 2

    Real3 normal = normalize(cross(P1 - P0, P2 - P1));

    Real lambda0 = flex2D_BCEcoords_D[index].x;  // barycentric coordinate
    Real lambda1 = flex2D_BCEcoords_D[index].y;  // barycentric coordinate
    Real lambda2 = 1 - lambda0 - lambda1;        // barycentric coordinate
    Real z_val = flex2D_BCEcoords_D[index].z;    // off-face coordinate

    Real3 P = P0 * lambda0 + P1 * lambda1 + P2 * lambda2 + z_val * normal;  // BCE marker position
    Real3 V = V0 * lambda0 + V1 * lambda1 + V2 * lambda2;                   // BCE marker velocity

    Real h = posRadD[flex_index].w;
    posRadD[flex_index] = mR4(P, h);
    velMasD[flex_index] = V;
}

// =============================================================================

ChBce::ChBce(std::shared_ptr<SphMarkerDataD> sortedSphMarkers_D,
             std::shared_ptr<ProximityDataD> markersProximity_D,
             std::shared_ptr<FsiData> fsiData,
             std::shared_ptr<SimParams> paramsH,
             std::shared_ptr<ChCounters> numObjects,
             bool verbose)
    : ChFsiBase(paramsH, numObjects),
      m_sortedSphMarkersD(sortedSphMarkers_D),
      m_markersProximityD(markersProximity_D),
      m_fsiData(fsiData),
      m_verbose(verbose) {
    m_totalForceRigid.resize(0);
    m_totalTorqueRigid.resize(0);
}

ChBce::~ChBce() {}

// -----------------------------------------------------------------------------

void ChBce::Initialize(std::shared_ptr<SphMarkerDataD> sphMarkers_D,
                       std::shared_ptr<FsiBodyStateD> fsiBodyState_D,
                       std::shared_ptr<FsiMeshStateD> fsiMesh1DState_D,
                       std::shared_ptr<FsiMeshStateD> fsiMesh2DState_D,
                       std::vector<int> fsiBodyBceNum) {
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), paramsH.get(), sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjectsH.get(), sizeof(ChCounters));
    CopyParams_NumberOfObjects(paramsH, numObjectsH);

    // Resizing the arrays used to modify the BCE velocity and pressure according to ADAMI
    m_totalForceRigid.resize(numObjectsH->numRigidBodies);
    m_totalTorqueRigid.resize(numObjectsH->numRigidBodies);

    int haveGhost = (numObjectsH->numGhostMarkers > 0) ? 1 : 0;
    int haveHelper = (numObjectsH->numHelperMarkers > 0) ? 1 : 0;
    int haveRigid = (numObjectsH->numRigidBodies > 0) ? 1 : 0;
    int haveFlex1D = (numObjectsH->numFlexBodies1D > 0) ? 1 : 0;
    int haveFlex2D = (numObjectsH->numFlexBodies2D > 0) ? 1 : 0;

    int num = haveHelper + haveGhost + haveRigid + haveFlex1D + haveFlex2D + 1;
    int numFlexRigidBoundaryMarkers =
        m_fsiData->referenceArray[num].y - m_fsiData->referenceArray[haveHelper + haveGhost].y;

    if (m_verbose) {
        printf("Total number of BCE particles = %d\n", numFlexRigidBoundaryMarkers);
        if (paramsH->bceType == BceVersion::ADAMI)
            printf("Boundary condition for rigid and flexible body is: ADAMI\n");
        if (paramsH->bceType == BceVersion::ORIGINAL)
            printf("Boundary condition for rigid and flexible body is: ORIGINAL\n");
        if (paramsH->bceTypeWall == BceVersion::ADAMI)
            printf("Boundary condition for fixed wall is: ADAMI\n");
        if (paramsH->bceTypeWall == BceVersion::ORIGINAL)
            printf("Boundary condition for fixed wall is: ORIGINAL\n");
    }

    auto numAllBce = numObjectsH->numBoundaryMarkers + numObjectsH->numRigidMarkers +  //
                     numObjectsH->numFlexMarkers1D + numObjectsH->numFlexMarkers2D;
    if ((int)numAllBce != numFlexRigidBoundaryMarkers) {
        throw std::runtime_error(
            "Error! number of flex and rigid and "
            "boundary markers are saved incorrectly!\n");
    }
    velMas_ModifiedBCE.resize(numFlexRigidBoundaryMarkers);
    rhoPreMu_ModifiedBCE.resize(numFlexRigidBoundaryMarkers);
    tauXxYyZz_ModifiedBCE.resize(numFlexRigidBoundaryMarkers);
    tauXyXzYz_ModifiedBCE.resize(numFlexRigidBoundaryMarkers);

    // Populate local position of BCE markers - on rigid bodies
    if (haveRigid) {
        Populate_RigidSPH_MeshPos_LRF(sphMarkers_D, fsiBodyState_D, fsiBodyBceNum);
        UpdateBodyMarkerState(sphMarkers_D, fsiBodyState_D);
    }

    // Populate local position of BCE markers - on flexible bodies
    if (haveFlex1D) {
        m_fsiData->flex1D_Nodes_D = m_fsiData->flex1D_Nodes_H;
        m_fsiData->flex1D_BCEsolids_D = m_fsiData->flex1D_BCEsolids_H;
        m_fsiData->flex1D_BCEcoords_D = m_fsiData->flex1D_BCEcoords_H;
        UpdateMeshMarker1DState(sphMarkers_D, fsiMesh1DState_D);
    }

    if (haveFlex2D) {
        m_fsiData->flex2D_Nodes_D = m_fsiData->flex2D_Nodes_H;
        m_fsiData->flex2D_BCEsolids_D = m_fsiData->flex2D_BCEsolids_H;
        m_fsiData->flex2D_BCEcoords_D = m_fsiData->flex2D_BCEcoords_H;
        UpdateMeshMarker2DState(sphMarkers_D, fsiMesh2DState_D);
    }
}

// -----------------------------------------------------------------------------

void ChBce::Populate_RigidSPH_MeshPos_LRF(std::shared_ptr<SphMarkerDataD> sphMarkers_D,
                                          std::shared_ptr<FsiBodyStateD> fsiBodyState_D,
                                          std::vector<int> fsiBodyBceNum) {
    // Create map between a BCE on a rigid body and the associated body ID
    uint start_bce = 0;
    for (int irigid = 0; irigid < fsiBodyBceNum.size(); irigid++) {
        uint end_bce = start_bce + fsiBodyBceNum[irigid];
        thrust::fill(m_fsiData->rigid_BCEsolids_D.begin() + start_bce, m_fsiData->rigid_BCEsolids_D.begin() + end_bce,
                     irigid);
        start_bce = end_bce;
    }

    uint nBlocks, nThreads;
    computeGridSize((uint)numObjectsH->numRigidMarkers, 256, nBlocks, nThreads);

    Populate_RigidSPH_MeshPos_LRF_D<<<nBlocks, nThreads>>>(
        mR3CAST(m_fsiData->rigid_BCEcoords_D), mR4CAST(sphMarkers_D->posRadD), U1CAST(m_fsiData->rigid_BCEsolids_D),
        mR3CAST(fsiBodyState_D->pos), mR4CAST(fsiBodyState_D->rot));

    hipDeviceSynchronize();
    cudaCheckError();
}

// -----------------------------------------------------------------------------

void ChBce::ReCalcVelocityPressureStress_BCE(thrust::device_vector<Real3>& velMas_ModifiedBCE,
                                             thrust::device_vector<Real4>& rhoPreMu_ModifiedBCE,
                                             thrust::device_vector<Real3>& tauXxYyZz_ModifiedBCE,
                                             thrust::device_vector<Real3>& tauXyXzYz_ModifiedBCE,
                                             const thrust::device_vector<Real4>& sortedPosRad,
                                             const thrust::device_vector<Real3>& sortedVelMas,
                                             const thrust::device_vector<Real4>& sortedRhoPreMu,
                                             const thrust::device_vector<Real3>& sortedTauXxYyZz,
                                             const thrust::device_vector<Real3>& sortedTauXyXzYz,
                                             const thrust::device_vector<uint>& cellStart,
                                             const thrust::device_vector<uint>& cellEnd,
                                             const thrust::device_vector<uint>& mapOriginalToSorted,
                                             const thrust::device_vector<uint>& extendedActivityIdD,
                                             const thrust::device_vector<Real3>& bceAcc,
                                             int4 updatePortion) {
    bool *isErrorH, *isErrorD;
    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

    // thread per particle
    int2 newPortion = mI2(updatePortion.x, updatePortion.w);
    if (paramsH->bceTypeWall == BceVersion::ORIGINAL) {
        // Only implement ADAMI BC for rigid body boundary.
        // Implement a simple BC for fixed wall to avoid unnecessary cost.
        newPortion = mI2(updatePortion.y, updatePortion.w);
    }
    uint numBCE = newPortion.y - newPortion.x;
    uint numThreads, numBlocks;
    computeGridSize(numBCE, 256, numBlocks, numThreads);

    BCE_VelocityPressureStress<<<numBlocks, numThreads>>>(
        mR3CAST(velMas_ModifiedBCE), mR4CAST(rhoPreMu_ModifiedBCE), mR3CAST(tauXxYyZz_ModifiedBCE),
        mR3CAST(tauXyXzYz_ModifiedBCE), mR4CAST(sortedPosRad), mR3CAST(sortedVelMas), mR4CAST(sortedRhoPreMu),
        mR3CAST(sortedTauXxYyZz), mR3CAST(sortedTauXyXzYz), U1CAST(cellStart), U1CAST(cellEnd),
        U1CAST(mapOriginalToSorted), U1CAST(extendedActivityIdD), mR3CAST(bceAcc), newPortion, isErrorD);

    hipDeviceSynchronize();
    cudaCheckError();

    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true)
        throw std::runtime_error("Error! program crashed in new_BCE_VelocityPressure!\n");

    hipFree(isErrorD);
    free(isErrorH);
}

//--------------------------------------------------------------------------------------------------------------------------------

void ChBce::CalcRigidBceAcceleration(thrust::device_vector<Real3>& bceAcc,
                                     const thrust::device_vector<Real4>& q_fsiBodies_D,
                                     const thrust::device_vector<Real3>& accRigid_fsiBodies_D,
                                     const thrust::device_vector<Real3>& omegaVelLRF_fsiBodies_D,
                                     const thrust::device_vector<Real3>& omegaAccLRF_fsiBodies_D,
                                     const thrust::device_vector<Real3>& rigid_BCEcoords_D,
                                     const thrust::device_vector<uint>& rigid_BCEsolids_D) {
    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize((uint)numObjectsH->numRigidMarkers, 256, numBlocks, numThreads);

    CalcRigidBceAccelerationD<<<numBlocks, numThreads>>>(
        mR3CAST(bceAcc), mR4CAST(q_fsiBodies_D), mR3CAST(accRigid_fsiBodies_D), mR3CAST(omegaVelLRF_fsiBodies_D),
        mR3CAST(omegaAccLRF_fsiBodies_D), mR3CAST(rigid_BCEcoords_D), U1CAST(rigid_BCEsolids_D));

    hipDeviceSynchronize();
    cudaCheckError();
}

void ChBce::CalcMeshMarker1DAcceleration(thrust::device_vector<Real3>& bceAcc,
                                         std::shared_ptr<FsiMeshStateD> fsiMesh1DState_D) {
    if (numObjectsH->numFlexBodies1D == 0)
        return;

    uint nBlocks, nThreads;
    computeGridSize((int)numObjectsH->numFlexMarkers1D, 256, nBlocks, nThreads);

    CalcMeshMarker1DAcceleration_D<<<nBlocks, nThreads>>>(  //
        mR3CAST(bceAcc),                                    //
        mR3CAST(fsiMesh1DState_D->acc_fsi_fea_D),           //
        U2CAST(m_fsiData->flex1D_Nodes_D),                  //
        U3CAST(m_fsiData->flex1D_BCEsolids_D),              //
        mR3CAST(m_fsiData->flex1D_BCEcoords_D)              //
    );
}

void ChBce::CalcMeshMarker2DAcceleration(thrust::device_vector<Real3>& bceAcc,
                                         std::shared_ptr<FsiMeshStateD> fsiMesh2DState_D) {
    if (numObjectsH->numFlexBodies2D == 0)
        return;

    uint nBlocks, nThreads;
    computeGridSize((int)numObjectsH->numFlexMarkers2D, 256, nBlocks, nThreads);

    CalcMeshMarker2DAcceleration_D<<<nBlocks, nThreads>>>(  //
        mR3CAST(bceAcc),                                    //
        mR3CAST(fsiMesh2DState_D->acc_fsi_fea_D),           //
        U3CAST(m_fsiData->flex2D_Nodes_D),                  //
        U3CAST(m_fsiData->flex2D_BCEsolids_D),              //
        mR3CAST(m_fsiData->flex2D_BCEcoords_D)              //
    );
}

// -----------------------------------------------------------------------------

void ChBce::ModifyBceVelocityPressureStress(std::shared_ptr<SphMarkerDataD> sphMarkers_D,
                                            std::shared_ptr<FsiBodyStateD> fsiBodyState_D,
                                            std::shared_ptr<FsiMeshStateD> fsiMesh1DState_D,
                                            std::shared_ptr<FsiMeshStateD> fsiMesh2DState_D) {
    auto size_ref = m_fsiData->referenceArray.size();
    auto numBceMarkers = m_fsiData->referenceArray[size_ref - 1].y - m_fsiData->referenceArray[0].y;

    auto N_solid = numObjectsH->numRigidMarkers + numObjectsH->numFlexMarkers1D + numObjectsH->numFlexMarkers2D;
    auto N_all = N_solid + numObjectsH->numBoundaryMarkers;

    if ((int)N_all != numBceMarkers) {
        throw std::runtime_error(
            "Error! Number of rigid, flexible and boundary markers are "
            "saved incorrectly. Thrown from ModifyBceVelocityPressureStress!\n");
    }

    if (!(velMas_ModifiedBCE.size() == numBceMarkers && rhoPreMu_ModifiedBCE.size() == numBceMarkers &&
          tauXxYyZz_ModifiedBCE.size() == numBceMarkers && tauXyXzYz_ModifiedBCE.size() == numBceMarkers)) {
        throw std::runtime_error(
            "Error! Size error velMas_ModifiedBCE and "
            "tauXxYyZz_ModifiedBCE and tauXyXzYz_ModifiedBCE and "
            "rhoPreMu_ModifiedBCE. Thrown from ModifyBceVelocityPressureStress!\n");
    }

    // Update portion set to boundary, rigid, and flexible BCE particles
    int4 updatePortion = mI4(m_fsiData->referenceArray[0].y, m_fsiData->referenceArray[1].y,
                             m_fsiData->referenceArray[2].y, m_fsiData->referenceArray[3].y);

    // Only update boundary BCE particles if no rigid/flexible particles
    if (size_ref == 2) {
        updatePortion.z = m_fsiData->referenceArray[1].y;
        updatePortion.w = m_fsiData->referenceArray[1].y;
    }

    // Update boundary and rigid/flexible BCE particles
    if (size_ref == 3)
        updatePortion.w = m_fsiData->referenceArray[2].y;

    if (paramsH->bceType == BceVersion::ADAMI) {
        // ADAMI boundary condition (wall, rigid, flexible)

        // Calculate the acceleration of rigid/flexible BCE particles if exist, used for ADAMI BC
        thrust::device_vector<Real3> bceAcc(N_solid);

        // Acceleration of rigid BCE particles
        if (numObjectsH->numRigidMarkers > 0) {
            CalcRigidBceAcceleration(bceAcc, fsiBodyState_D->rot, fsiBodyState_D->lin_acc, fsiBodyState_D->ang_vel,
                                     fsiBodyState_D->ang_acc, m_fsiData->rigid_BCEcoords_D,
                                     m_fsiData->rigid_BCEsolids_D);
        }
        // Acceleration of flexible BCE particles
        if (numObjectsH->numFlexMarkers1D > 0) {
            CalcMeshMarker1DAcceleration(bceAcc, fsiMesh1DState_D);
        }
        if (numObjectsH->numFlexMarkers2D > 0) {
            CalcMeshMarker1DAcceleration(bceAcc, fsiMesh2DState_D);
        }

        if (paramsH->bceTypeWall == BceVersion::ORIGINAL) {
            // ADAMI BC for rigid/flexible body, ORIGINAL BC for fixed wall
            thrust::copy(sphMarkers_D->velMasD.begin() + updatePortion.x,
                         sphMarkers_D->velMasD.begin() + updatePortion.y, velMas_ModifiedBCE.begin());
            thrust::copy(sphMarkers_D->rhoPresMuD.begin() + updatePortion.x,
                         sphMarkers_D->rhoPresMuD.begin() + updatePortion.y, rhoPreMu_ModifiedBCE.begin());
            if (paramsH->elastic_SPH) {
                thrust::copy(sphMarkers_D->tauXxYyZzD.begin() + updatePortion.x,
                             sphMarkers_D->tauXxYyZzD.begin() + updatePortion.y, tauXxYyZz_ModifiedBCE.begin());
                thrust::copy(sphMarkers_D->tauXyXzYzD.begin() + updatePortion.x,
                             sphMarkers_D->tauXyXzYzD.begin() + updatePortion.y, tauXyXzYz_ModifiedBCE.begin());
            }
            if (N_solid > 0) {
                ReCalcVelocityPressureStress_BCE(
                    velMas_ModifiedBCE, rhoPreMu_ModifiedBCE, tauXxYyZz_ModifiedBCE, tauXyXzYz_ModifiedBCE,
                    m_sortedSphMarkersD->posRadD, m_sortedSphMarkersD->velMasD, m_sortedSphMarkersD->rhoPresMuD,
                    m_sortedSphMarkersD->tauXxYyZzD, m_sortedSphMarkersD->tauXyXzYzD, m_markersProximityD->cellStartD,
                    m_markersProximityD->cellEndD, m_markersProximityD->mapOriginalToSorted,
                    m_fsiData->extendedActivityIdD, bceAcc, updatePortion);
            }
        } else if (paramsH->bceTypeWall == BceVersion::ADAMI) {
            // ADAMI BC for both rigid/flexible body and fixed wall

            ReCalcVelocityPressureStress_BCE(
                velMas_ModifiedBCE, rhoPreMu_ModifiedBCE, tauXxYyZz_ModifiedBCE, tauXyXzYz_ModifiedBCE,
                m_sortedSphMarkersD->posRadD, m_sortedSphMarkersD->velMasD, m_sortedSphMarkersD->rhoPresMuD,
                m_sortedSphMarkersD->tauXxYyZzD, m_sortedSphMarkersD->tauXyXzYzD, m_markersProximityD->cellStartD,
                m_markersProximityD->cellEndD, m_markersProximityD->mapOriginalToSorted, m_fsiData->extendedActivityIdD,
                bceAcc, updatePortion);
        }

        bceAcc.clear();
    } else {
        // ORIGINAL boundary condition for all boundaries (wall, rigid, flexible)

        thrust::copy(sphMarkers_D->velMasD.begin() + updatePortion.x, sphMarkers_D->velMasD.begin() + updatePortion.w,
                     velMas_ModifiedBCE.begin());
        thrust::copy(sphMarkers_D->rhoPresMuD.begin() + updatePortion.x,
                     sphMarkers_D->rhoPresMuD.begin() + updatePortion.w, rhoPreMu_ModifiedBCE.begin());
        if (paramsH->elastic_SPH) {
            thrust::copy(sphMarkers_D->tauXxYyZzD.begin() + updatePortion.x,
                         sphMarkers_D->tauXxYyZzD.begin() + updatePortion.w, tauXxYyZz_ModifiedBCE.begin());
            thrust::copy(sphMarkers_D->tauXyXzYzD.begin() + updatePortion.x,
                         sphMarkers_D->tauXyXzYzD.begin() + updatePortion.w, tauXyXzYz_ModifiedBCE.begin());
        }
    }
}

// -----------------------------------------------------------------------------

void ChBce::Rigid_Forces_Torques(std::shared_ptr<SphMarkerDataD> sphMarkers_D,
                                 std::shared_ptr<FsiBodyStateD> fsiBodyState_D) {
    if (numObjectsH->numRigidBodies == 0)
        return;

    thrust::fill(m_fsiData->rigid_FSI_ForcesD.begin(), m_fsiData->rigid_FSI_ForcesD.end(), mR3(0));
    thrust::fill(m_fsiData->rigid_FSI_TorquesD.begin(), m_fsiData->rigid_FSI_TorquesD.end(), mR3(0));

    uint nBlocks, nThreads;
    computeGridSize((uint)numObjectsH->numRigidMarkers, 256, nBlocks, nThreads);

    CalcRigidForces_D<<<nBlocks, nThreads>>>(
        mR3CAST(m_fsiData->rigid_FSI_ForcesD), mR3CAST(m_fsiData->rigid_FSI_TorquesD), mR4CAST(m_fsiData->derivVelRhoD),
        mR4CAST(m_fsiData->derivVelRhoD_old), mR4CAST(sphMarkers_D->posRadD), U1CAST(m_fsiData->rigid_BCEsolids_D),
        mR3CAST(fsiBodyState_D->pos), mR3CAST(m_fsiData->rigid_BCEcoords_D));

    hipDeviceSynchronize();
    cudaCheckError();
}

void ChBce::Flex1D_Forces(std::shared_ptr<SphMarkerDataD> sphMarkers_D,
                          std::shared_ptr<FsiMeshStateD> fsiMesh1DState_D) {
    if (numObjectsH->numFlexBodies1D == 0)
        return;

    // Initialize accumulator to zero
    thrust::fill(m_fsiData->flex1D_FSIforces_D.begin(), m_fsiData->flex1D_FSIforces_D.end(), mR3(0));

    uint nBlocks, nThreads;
    computeGridSize((int)numObjectsH->numFlexMarkers1D, 256, nBlocks, nThreads);

    CalcFlex1DForces_D<<<nBlocks, nThreads>>>(                                   //
        mR3CAST(m_fsiData->flex1D_FSIforces_D),                                  //
        mR4CAST(m_fsiData->derivVelRhoD), mR4CAST(m_fsiData->derivVelRhoD_old),  //
        U2CAST(m_fsiData->flex1D_Nodes_D),                                       //
        U3CAST(m_fsiData->flex1D_BCEsolids_D),                                   //
        mR3CAST(m_fsiData->flex1D_BCEcoords_D)                                   //
    );

    hipDeviceSynchronize();
    cudaCheckError();
}

void ChBce::Flex2D_Forces(std::shared_ptr<SphMarkerDataD> sphMarkers_D,
                          std::shared_ptr<FsiMeshStateD> fsiMesh2DState_D) {
    if (numObjectsH->numFlexBodies2D == 0)
        return;

    // Initialize accumulator to zero
    thrust::fill(m_fsiData->flex2D_FSIforces_D.begin(), m_fsiData->flex2D_FSIforces_D.end(), mR3(0));

    uint nBlocks, nThreads;
    computeGridSize((int)numObjectsH->numFlexMarkers2D, 256, nBlocks, nThreads);

    CalcFlex2DForces_D<<<nBlocks, nThreads>>>(                                   //
        mR3CAST(m_fsiData->flex2D_FSIforces_D),                                  //
        mR4CAST(m_fsiData->derivVelRhoD), mR4CAST(m_fsiData->derivVelRhoD_old),  //
        U3CAST(m_fsiData->flex2D_Nodes_D),                                       //
        U3CAST(m_fsiData->flex2D_BCEsolids_D),                                   //
        mR3CAST(m_fsiData->flex2D_BCEcoords_D)                                   //
    );

    hipDeviceSynchronize();
    cudaCheckError();
}

// -----------------------------------------------------------------------------

void ChBce::UpdateBodyMarkerState(std::shared_ptr<SphMarkerDataD> sphMarkers_D,
                                  std::shared_ptr<FsiBodyStateD> fsiBodyState_D) {
    if (numObjectsH->numRigidBodies == 0)
        return;

    uint nBlocks, nThreads;
    computeGridSize((int)numObjectsH->numRigidMarkers, 256, nBlocks, nThreads);

    UpdateBodyMarkerStateD<<<nBlocks, nThreads>>>(
        mR4CAST(sphMarkers_D->posRadD), mR3CAST(sphMarkers_D->velMasD), mR3CAST(m_fsiData->rigid_BCEcoords_D),
        U1CAST(m_fsiData->rigid_BCEsolids_D), mR3CAST(fsiBodyState_D->pos), mR4CAST(fsiBodyState_D->lin_vel),
        mR3CAST(fsiBodyState_D->ang_vel), mR4CAST(fsiBodyState_D->rot));

    hipDeviceSynchronize();
    cudaCheckError();
}

void ChBce::UpdateMeshMarker1DState(std::shared_ptr<SphMarkerDataD> sphMarkers_D,
                                    std::shared_ptr<FsiMeshStateD> fsiMesh1DState_D) {
    if (numObjectsH->numFlexBodies1D == 0)
        return;

    uint nBlocks, nThreads;
    computeGridSize((int)numObjectsH->numFlexMarkers1D, 256, nBlocks, nThreads);

    UpdateMeshMarker1DState_D<<<nBlocks, nThreads>>>(                                        //
        mR4CAST(sphMarkers_D->posRadD), mR3CAST(sphMarkers_D->velMasD),                      //
        mR3CAST(fsiMesh1DState_D->pos_fsi_fea_D), mR3CAST(fsiMesh1DState_D->vel_fsi_fea_D),  //
        U2CAST(m_fsiData->flex1D_Nodes_D),                                                   //
        U3CAST(m_fsiData->flex1D_BCEsolids_D),                                               //
        mR3CAST(m_fsiData->flex1D_BCEcoords_D)                                               //
    );

    hipDeviceSynchronize();
    cudaCheckError();
}

void ChBce::UpdateMeshMarker2DState(std::shared_ptr<SphMarkerDataD> sphMarkers_D,
                                    std::shared_ptr<FsiMeshStateD> fsiMesh2DState_D) {
    if (numObjectsH->numFlexBodies2D == 0)
        return;

    uint nBlocks, nThreads;
    computeGridSize((int)numObjectsH->numFlexMarkers2D, 256, nBlocks, nThreads);

    UpdateMeshMarker2DState_D<<<nBlocks, nThreads>>>(                                        //
        mR4CAST(sphMarkers_D->posRadD), mR3CAST(sphMarkers_D->velMasD),                      //
        mR3CAST(fsiMesh2DState_D->pos_fsi_fea_D), mR3CAST(fsiMesh2DState_D->vel_fsi_fea_D),  //
        U3CAST(m_fsiData->flex2D_Nodes_D),                                                   //
        U3CAST(m_fsiData->flex2D_BCEsolids_D),                                               //
        mR3CAST(m_fsiData->flex2D_BCEcoords_D)                                               //
    );

    hipDeviceSynchronize();
    cudaCheckError();
}

}  // end namespace fsi
}  // end namespace chrono
